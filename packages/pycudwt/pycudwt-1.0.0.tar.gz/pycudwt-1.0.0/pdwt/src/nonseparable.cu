#include "hip/hip_runtime.h"
#include "nonseparable.h"
#include "common.h"

#ifdef SEPARATE_COMPILATION
// Required for separate compilation (see Makefile)
#ifndef CONSTMEM_FILTERS_NS
#define CONSTMEM_FILTERS_NS
__constant__ DTYPE c_kern_LL[MAX_FILTER_WIDTH * MAX_FILTER_WIDTH];
__constant__ DTYPE c_kern_LH[MAX_FILTER_WIDTH * MAX_FILTER_WIDTH];
__constant__ DTYPE c_kern_HL[MAX_FILTER_WIDTH * MAX_FILTER_WIDTH];
__constant__ DTYPE c_kern_HH[MAX_FILTER_WIDTH * MAX_FILTER_WIDTH];
#endif
#endif

// outer product of arrays "a", "b" of length "len"
DTYPE* w_outer(DTYPE* a, DTYPE* b, int len) {
    DTYPE* res = (DTYPE*) calloc(len*len, sizeof(DTYPE));
    for (int i = 0; i < len; i++) {
        for (int j = 0; j < len; j++) {
            res[i*len+j] = a[i]*b[j];
        }
    }
    return res;
}


/// Compute the four filters A, H, V, D  from a family name.
/// These filters are separable, i.e computed from 1D filters.
/// wname: name of the filter ("haar", "db3", "sym4", ...)
/// direction: 1 for forward transform, -1 for inverse transform
/// Returns : the filter width "hlen" if success ; a negative value otherwise.
int w_compute_filters(const char* wname, int direction, int do_swt) {
    if (direction == 0) {
        puts("ERROR: w_compute_filters(): please specify a direction for second argument : +1 for forward, -1 for inverse)");
        return -1;
    }
    int hlen = 0;
    DTYPE* f1_l; // 1D lowpass
    DTYPE* f1_h; // 1D highpass
    DTYPE* f2_a, *f2_h, *f2_v, *f2_d; // 2D filters

    // Haar filters has specific kernels
    if (!do_swt) {
        if ((!strcasecmp(wname, "haar")) || (!strcasecmp(wname, "db1")) || (!strcasecmp(wname, "bior1.1")) || (!strcasecmp(wname, "rbior1.1"))) {
            return 2;
        }
    }

    // Browse available filters (see filters.h)
    int i;
    for (i = 0; i < 72; i++) {
        if (!strcasecmp(wname, all_filters[i].wname)) {
            hlen = all_filters[i].hlen;
            if (direction > 0) {
                f1_l = all_filters[i].f_l;
                f1_h = all_filters[i].f_h;
            }
            else {
                f1_l = all_filters[i].i_l;
                f1_h = all_filters[i].i_h;
            }
            break;
        }
    }
    if (hlen == 0) {
        printf("ERROR: w_compute_filters(): unknown filter %s\n", wname);
        return -2;
    }

    // Create the separable 2D filters
    f2_a = w_outer(f1_l, f1_l, hlen);
    f2_h = w_outer(f1_l, f1_h, hlen); // CHECKME
    f2_v = w_outer(f1_h, f1_l, hlen);
    f2_d = w_outer(f1_h, f1_h, hlen);

    // Copy the filters to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_LL), f2_a, hlen*hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_LH), f2_h, hlen*hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice); // CHECKME
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_HL), f2_v, hlen*hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_kern_HH), f2_d, hlen*hlen*sizeof(DTYPE), 0, hipMemcpyHostToDevice);

    return hlen;
}


int w_set_filters_forward_nonseparable(DTYPE* filter1, DTYPE* filter2, DTYPE* filter3, DTYPE* filter4, uint len) {
    if (hipMemcpyToSymbol(HIP_SYMBOL(c_kern_LL), filter1, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess
            || hipMemcpyToSymbol(HIP_SYMBOL(c_kern_LH), filter2, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess
            || hipMemcpyToSymbol(HIP_SYMBOL(c_kern_HL), filter3, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess
            || hipMemcpyToSymbol(HIP_SYMBOL(c_kern_HH), filter4, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess)
    {
        return -3;
    }
    return 0;
}

int w_set_filters_inverse_nonseparable(DTYPE* filter1, DTYPE* filter2, DTYPE* filter3, DTYPE* filter4, uint len) {
    if (hipMemcpyToSymbol(HIP_SYMBOL(c_kern_LL), filter1, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess
        || hipMemcpyToSymbol(HIP_SYMBOL(c_kern_LH), filter2, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess
        || hipMemcpyToSymbol(HIP_SYMBOL(c_kern_HL), filter3, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess
        || hipMemcpyToSymbol(HIP_SYMBOL(c_kern_HH), filter4, len*len*sizeof(DTYPE), 0, hipMemcpyHostToDevice) != hipSuccess)
    {
        return -3;
    }
    return 0;
}






// must be run with grid size = (Nc/2, Nr/2)  where Nr = numrows of input image
__global__ void w_kern_forward(DTYPE* img, DTYPE* c_a, DTYPE* c_h, DTYPE* c_v, DTYPE* c_d, int Nr, int Nc, int hlen) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;

    int Nr_is_odd = (Nr & 1);
    int Nr2 = (Nr + Nr_is_odd)/2;
    int Nc_is_odd = (Nc & 1);
    int Nc2 = (Nc + Nc_is_odd)/2;

    if (gidy < Nr2 && gidx < Nc2) {
        int c, hL, hR;
        if (hlen & 1) { // odd kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even kernel size : center is shifted to the left
            c = hlen/2 - 1;
            hL = c;
            hR = c+1;
        }
        DTYPE res_a = 0, res_h = 0, res_v = 0, res_d = 0;
        DTYPE img_val;

        // Convolution with periodic boundaries extension.
        // The following can be sped-up by splitting into 3*3 loops, but it would be a nightmare for readability
        for (int jy = 0; jy <= hR+hL; jy++) {
            int idx_y = gidy*2 - c + jy;
            if (idx_y < 0) idx_y += (Nr + Nr_is_odd); // if N is odd, image is virtually extended
            // no "else if", since idx_y can be > N-1  after being incremented
            if (idx_y > Nr-1) {
                if ((idx_y == Nr) && (Nr_is_odd)) idx_y--; // if N is odd, repeat the right-most element
                else idx_y -= (Nr + Nr_is_odd); // if N is odd, image is virtually extended
            }

            for (int jx = 0; jx <= hR+hL; jx++) {
                int idx_x = gidx*2 - c + jx;
                if (idx_x < 0) idx_x += (Nc + Nc_is_odd); // if N is odd, image is virtually extended
                // no "else if", since idx_x can be > N-1  after being incremented
                if (idx_x > Nc-1) {
                    if ((idx_x == Nc) && (Nc_is_odd)) idx_x--; // if N is odd, repeat the right-most element
                    else idx_x -= (Nc + Nc_is_odd); // if N is odd, image is virtually extended
                }

                img_val = img[idx_y*Nc + idx_x];
                res_a += img_val * c_kern_LL[(hlen-1-jy)*hlen + (hlen-1 - jx)];
                res_h += img_val * c_kern_LH[(hlen-1-jy)*hlen + (hlen-1 - jx)];
                res_v += img_val * c_kern_HL[(hlen-1-jy)*hlen + (hlen-1 - jx)];
                res_d += img_val * c_kern_HH[(hlen-1-jy)*hlen + (hlen-1 - jx)];
            }
        }
        c_a[gidy* Nc2 + gidx] = res_a;
        c_h[gidy* Nc2 + gidx] = res_h;
        c_v[gidy* Nc2 + gidx] = res_v;
        c_d[gidy* Nc2 + gidx] = res_d;
    }
}




// must be run with grid size = (2*Nr, 2*Nc) ; Nr = numrows of input
__global__ void w_kern_inverse(DTYPE* img, DTYPE* c_a, DTYPE* c_h, DTYPE* c_v, DTYPE* c_d, int Nr, int Nc, int Nr2, int Nc2, int hlen) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr2 && gidx < Nc2) {

        int c, hL, hR;
        int hlen2 = hlen/2; // Convolutions with even/odd indices of the kernels
        if (hlen2 & 1) { // odd half-kernel size
            c = hlen2/2;
            hL = c;
            hR = c;
        }
        else { // even half-kernel size : center is shifted to the RIGHT for reconstruction.
            c = hlen2/2 - 0;
            hL = c;
            hR = c-1;
            // virtual id for shift
            // TODO : for the very first convolution (on the edges), this is not exactly accurate (?)
            gidx += 1;
            gidy += 1;
        }
        int jy1 = c - gidy/2;
        int jy2 = Nr - 1 - gidy/2 + c;
        int jx1 = c - gidx/2;
        int jx2 = Nc - 1 - gidx/2 + c;

        // There are 4 threads/coeff index. Each thread will do a convolution with the even/odd indices of the kernels along each dimension.
        int offset_x = 1-(gidx & 1);
        int offset_y = 1-(gidy & 1);

        DTYPE res_a = 0, res_h = 0, res_v = 0, res_d = 0;
        for (int jy = 0; jy <= hR+hL; jy++) {
            int idx_y = gidy/2 - c + jy;
            if (jy < jy1) idx_y += Nr;
            if (jy > jy2) idx_y -= Nr;
            for (int jx = 0; jx <= hR+hL; jx++) {
                int idx_x = gidx/2 - c + jx;
                if (jx < jx1) idx_x += Nc;
                if (jx > jx2) idx_x -= Nc;

                res_a += c_a[idx_y*Nc + idx_x] * c_kern_LL[(hlen-1- (2*jy + offset_y))*hlen + (hlen-1 - (2*jx + offset_x))];
                res_h += c_h[idx_y*Nc + idx_x] * c_kern_LH[(hlen-1- (2*jy + offset_y))*hlen + (hlen-1 - (2*jx + offset_x))];
                res_v += c_v[idx_y*Nc + idx_x] * c_kern_HL[(hlen-1- (2*jy + offset_y))*hlen + (hlen-1 - (2*jx + offset_x))];
                res_d += c_d[idx_y*Nc + idx_x] * c_kern_HH[(hlen-1- (2*jy + offset_y))*hlen + (hlen-1 - (2*jx + offset_x))];
            }
        }
        if ((hlen2 & 1) == 1) img[gidy * Nc2 + gidx] = res_a + res_h + res_v + res_d;
        else img[(gidy-1) * Nc2 + (gidx-1)] = res_a + res_h + res_v + res_d;
    }
}







int w_forward(DTYPE* d_image, DTYPE** d_coeffs, DTYPE* d_tmp, w_info winfos) {

    int Nr = winfos.Nr, Nc = winfos.Nc, levels = winfos.nlevels, hlen = winfos.hlen;
    int tpb = 16; // TODO : tune for max perfs.
    int Nc2 = Nc, Nr2 = Nr;
    int Nc2_old = Nc2, Nr2_old = Nr2;
    w_div2(&Nr2); w_div2(&Nc2);
    DTYPE* d_tmp1, *d_tmp2;
    d_tmp1 = d_coeffs[0];
    d_tmp2 = d_tmp;

    // First level
    dim3 n_blocks = dim3(w_iDivUp(Nc2, tpb), w_iDivUp(Nr2, tpb), 1);
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    w_kern_forward<<<n_blocks, n_threads_per_block>>>(d_image, d_coeffs[0], d_coeffs[1], d_coeffs[2], d_coeffs[3], Nr, Nc, hlen);

    for (int i=1; i < levels; i++) {
        Nr2_old = Nr2; Nc2_old = Nc2;
        w_div2(&Nr2); w_div2(&Nc2);
        n_blocks = dim3(w_iDivUp(Nc2, tpb), w_iDivUp(Nr2, tpb), 1);
        w_kern_forward<<<n_blocks, n_threads_per_block>>>(d_tmp1, d_tmp2, d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], Nr2_old, Nc2_old, hlen);
        w_swap_ptr(&d_tmp1, &d_tmp2);
    }
    if ((levels > 1) && ((levels & 1) == 0)) hipMemcpy(d_coeffs[0], d_tmp, Nr2*Nc2*sizeof(DTYPE), hipMemcpyDeviceToDevice);
    return 0;
}


int w_inverse(DTYPE* d_image, DTYPE** d_coeffs, DTYPE* d_tmp, w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, levels = winfos.nlevels, hlen = winfos.hlen;
     // Table of sizes. FIXME: consider adding this in the w_info structure
    int tNr[levels+1]; tNr[0] = Nr;
    int tNc[levels+1]; tNc[0] = Nc;
    for (int i = 1; i <= levels; i++) {
        tNr[i] = tNr[i-1];
        tNc[i] = tNc[i-1];
        w_div2(tNr + i);
        w_div2(tNc + i);
    }
    int tpb = 16; // TODO : tune for max perfs.

    DTYPE* d_tmp1, *d_tmp2;
    d_tmp1 = d_coeffs[0];
    d_tmp2 = d_tmp;

    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    dim3 n_blocks;
    for (int i = levels-1; i >= 1; i--) {
        n_blocks = dim3(w_iDivUp(tNc[i], tpb), w_iDivUp(tNr[i], tpb), 1);
        w_kern_inverse<<<n_blocks, n_threads_per_block>>>(d_tmp2, d_tmp1, d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], tNr[i+1], tNc[i+1], tNr[i], tNc[i], hlen);
        w_swap_ptr(&d_tmp1, &d_tmp2);
    }
    if ((levels > 1) && ((levels & 1) == 0)) hipMemcpy(d_coeffs[0], d_tmp, tNr[1]*tNc[1]*sizeof(DTYPE), hipMemcpyDeviceToDevice);
    // First level
    n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    w_kern_inverse<<<n_blocks, n_threads_per_block>>>(d_image, d_coeffs[0], d_coeffs[1], d_coeffs[2], d_coeffs[3], tNr[1], tNc[1], Nr, Nc, hlen);

    return 0;
}





/// ----------------------------------------------------------------------------
/// -------------------------   Undecimated DWT --------------------------------
/// ----------------------------------------------------------------------------



// must be run with grid size = (Nc, Nr)  where Nr = numrows of input image
__global__ void w_kern_forward_swt(DTYPE* img, DTYPE* c_a, DTYPE* c_h, DTYPE* c_v, DTYPE* c_d, int Nr, int Nc, int hlen, int level) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr && gidx < Nc) {

        int factor = 1 << (level - 1);
        int c, hL, hR;
        if (hlen & 1) { // odd kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even kernel size : center is shifted to the left
            c = hlen/2 - 1;
            hL = c;
            hR = c+1;
        }

        c *= factor;
        int jx1 = c - gidx;
        int jx2 = Nc - 1 - gidx + c;
        int jy1 = c - gidy;
        int jy2 = Nr - 1 - gidy + c;
        DTYPE res_a = 0, res_h = 0, res_v = 0, res_d = 0;
        DTYPE img_val;

        // Convolution with periodic boundaries extension.
        // The filters are 2-upsampled at each level : [h0, h1, h2, h3] --> [h0, 0, h1, 0, h2, 0, h3, 0]
        for (int jy = 0; jy <= hR+hL; jy++) {
            int idx_y = gidy - c + factor*jy;
            if (factor*jy < jy1) idx_y += Nr;
            if (factor*jy > jy2) idx_y -= Nr;
            for (int jx = 0; jx <= hR+hL; jx++) {
                int idx_x = gidx + jx*factor - c;
                if (factor*jx < jx1) idx_x += Nc;
                if (factor*jx > jx2) idx_x -= Nc;

                img_val = img[idx_y*Nc + idx_x];
                res_a += img_val * c_kern_LL[(hlen-1-jy)*hlen + (hlen-1 - jx)];
                res_h += img_val * c_kern_LH[(hlen-1-jy)*hlen + (hlen-1 - jx)];
                res_v += img_val * c_kern_HL[(hlen-1-jy)*hlen + (hlen-1 - jx)];
                res_d += img_val * c_kern_HH[(hlen-1-jy)*hlen + (hlen-1 - jx)];

            }
        }
        c_a[gidy* Nc + gidx] = res_a;
        c_h[gidy* Nc + gidx] = res_h;
        c_v[gidy* Nc + gidx] = res_v;
        c_d[gidy* Nc + gidx] = res_d;
    }
}




// must be run with grid size = (2*Nr, 2*Nc) ; Nr = numrows of input
__global__ void w_kern_inverse_swt(DTYPE* img, DTYPE* c_a, DTYPE* c_h, DTYPE* c_v, DTYPE* c_d, int Nr, int Nc, int hlen, int level) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidy < Nr && gidx < Nc) {

        int factor = 1 << (level - 1);
        int c, hL, hR;
        if (hlen & 1) { // odd half-kernel size
            c = hlen/2;
            hL = c;
            hR = c;
        }
        else { // even half-kernel size : center is shifted to the RIGHT for reconstruction.
            c = hlen/2 - 0;
            hL = c;
            hR = c-1;
        }
        c *= factor;
        int jy1 = c - gidy;
        int jy2 = Nr - 1 - gidy + c;
        int jx1 = c - gidx;
        int jx2 = Nc - 1 - gidx + c;

        DTYPE res_a = 0, res_h = 0, res_v = 0, res_d = 0;
        for (int jy = 0; jy <= hR+hL; jy++) {
            int idx_y = gidy - c + jy*factor;
            if (factor*jy < jy1) idx_y += Nr;
            if (factor*jy > jy2) idx_y -= Nr;
            for (int jx = 0; jx <= hR+hL; jx++) {
                int idx_x = gidx - c + jx*factor;
                if (factor*jx < jx1) idx_x += Nc;
                if (factor*jx > jx2) idx_x -= Nc;

                res_a += c_a[idx_y*Nc + idx_x] * c_kern_LL[(hlen-1-jy)*hlen + (hlen-1 - jx)]/4;
                res_h += c_h[idx_y*Nc + idx_x] * c_kern_LH[(hlen-1-jy)*hlen + (hlen-1 - jx)]/4;
                res_v += c_v[idx_y*Nc + idx_x] * c_kern_HL[(hlen-1-jy)*hlen + (hlen-1 - jx)]/4;
                res_d += c_d[idx_y*Nc + idx_x] * c_kern_HH[(hlen-1-jy)*hlen + (hlen-1 - jx)]/4;
            }
        }
        img[gidy * Nc + gidx] = res_a + res_h + res_v + res_d;
    }
}






int w_forward_swt(DTYPE* d_image, DTYPE** d_coeffs, DTYPE* d_tmp, w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, levels = winfos.nlevels, hlen = winfos.hlen;

    DTYPE* d_tmp1, *d_tmp2;
    d_tmp1 = d_coeffs[0];
    d_tmp2 = d_tmp;

    // First level
    int tpb = 16; // TODO : tune for max perfs.
    dim3 n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    w_kern_forward_swt<<<n_blocks, n_threads_per_block>>>(d_image, d_coeffs[0], d_coeffs[1], d_coeffs[2], d_coeffs[3], Nr, Nc, hlen, 1);
    for (int i=1; i < levels; i++) {
        w_kern_forward_swt<<<n_blocks, n_threads_per_block>>>(d_tmp1, d_tmp2, d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], Nr, Nc, hlen, i+1);
        w_swap_ptr(&d_tmp1, &d_tmp2);
    }
    if ((levels & 1) == 0) hipMemcpy(d_coeffs[0], d_tmp, Nr*Nc*sizeof(DTYPE), hipMemcpyDeviceToDevice);
    return 0;
}



int w_inverse_swt(DTYPE* d_image, DTYPE** d_coeffs, DTYPE* d_tmp, w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, levels = winfos.nlevels, hlen = winfos.hlen;
    DTYPE* d_tmp1, *d_tmp2;
    d_tmp1 = d_coeffs[0];
    d_tmp2 = d_tmp;

    int tpb = 16; // TODO : tune for max perfs.
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    dim3 n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    for (int i = levels-1; i >= 1; i--) {
        w_kern_inverse_swt<<<n_blocks, n_threads_per_block>>>(d_tmp2, d_tmp1, d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], Nr, Nc, hlen, i+1);
        w_swap_ptr(&d_tmp1, &d_tmp2);
    }
    if ((levels & 1) == 0) hipMemcpy(d_coeffs[0], d_tmp, Nr*Nc*sizeof(DTYPE), hipMemcpyDeviceToDevice);
    // First scale
    n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    w_kern_inverse_swt<<<n_blocks, n_threads_per_block>>>(d_image, d_coeffs[0], d_coeffs[1], d_coeffs[2], d_coeffs[3], Nr, Nc, hlen, 1);

    return 0;
}



