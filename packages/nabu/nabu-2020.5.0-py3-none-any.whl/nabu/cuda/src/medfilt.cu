#include "hip/hip_runtime.h"
#include "boundary.h"
typedef unsigned int uint;

#ifndef MEDFILT_X
    #define MEDFILT_X 3
#endif
#ifndef MEDFILT_Y
    #define MEDFILT_Y 3
#endif


#ifndef DO_THRESHOLD
    #define DO_THRESHOLD 0
#endif


// General-purpose 2D (or batched 2D) median filter with a square footprint.
// Boundary handling is customized via the USED_CONV_MODE macro (see boundary.h)
// Most of the time is spent computing the median, so this kernel can be sped up by
//  - creating dedicated kernels for 3x3, 5x5 (see http://ndevilla.free.fr/median/median/src/optmed.c)
//  - Using a quickselect algorithm instead of sorting (see http://ndevilla.free.fr/median/median/src/quickselect.c)
__global__ void medfilt2d(
    float * input,
    float * output,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz,  // input/output depth
    float threshold // threshold for thresholded median filter
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(MEDFILT_X);

    // Get elements in a 3x3 neighborhood
    float elements[MEDFILT_X*MEDFILT_Y] = {0};
    for (int jy = 0; jy <= hR+hL; jy++) {
        CONV_IDX_Y; // Get index "y"
        for (int jx = 0; jx <= hR+hL; jx++) {
            CONV_IDX_X; // Get index "x"
            elements[jy*MEDFILT_Y+jx] = READ_IMAGE_2D_XY;
        }
    }
    // Sort the elements with insertion sort
    // TODO quickselect ?
    int i = 1, j;
    while (i < MEDFILT_X*MEDFILT_Y) {
        j = i;
        while (j > 0 && elements[j-1] > elements[j]) {
            float tmp = elements[j];
            elements[j] = elements[j-1];
            elements[j-1] = tmp;
            j--;
        }
        i++;
    }
    float median = elements[MEDFILT_X*MEDFILT_Y/2];

    #if DO_THRESHOLD == 1
    float out_val = 0.0f;
    uint idx = (gidz*Ny + gidy)*Nx + gidx;
    if (input[idx] >= median + threshold) out_val = median;
    else out_val = input[idx];
    output[idx] = out_val;
    #else
    output[(gidz*Ny + gidy)*Nx + gidx] = median;
    #endif
}

