
#include <hip/hip_runtime.h>
/*
    Perform a "half tomography" sinogram conversion.
    A 360 degrees sinogram is converted to a 180 degrees sinogram with a
    field of view extended (at most) twice".
    *
    Parameters:
    * sinogram: the 360 degrees sinogram, shape (n_angles, n_x)
    * output: the 160 degrees sinogram, shape (n_angles/2, rotation_axis_position * 2)
    * weights: an array of weight, size n_x - rotation_axis_position
*/
__global__ void halftomo_kernel(
    float* sinogram,
    float* output,
    float* weights,
    int n_angles,
    int n_x,
    int rotation_axis_position
) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int n_a2 = n_angles / 2;
    int d = n_x - rotation_axis_position;
    int n_x2  = 2 * rotation_axis_position;

    if ((x >= n_x2) || (y >= n_a2)) return;

    // output[:, :nx - d] = sino[:n_a2, :nx - d]
    if (x < n_x - d) {
        output[y * n_x2 + x] = sinogram[y * n_x + x];
    }



    // output[:, nx - d : nx] = (1 - weights) * sino[:n_a2, nx - d :]
    //                        + weights * sino[n_a2:, ::-1][:, d : 2 * d]
    else if (x < n_x) { // x in [n_x - d, n_x [
        // i in [nx - d - 1, nx - 2d - 1[  (down)
        // (n_x2 - 1 - x)    in   ] n_x2 - 1 - n_x,  n_x2 - 1 - n_x + d ]
        //                      = ] n_x - 2d - 1,  n_x - d - 1 ]  (up)
        float w = weights[x - (n_x - d)];
        output[y * n_x2 + x] = (1.0f - w) * sinogram[y*n_x + x] \
                                   + w * sinogram[(n_a2 + y)*n_x + (n_x2 - 1 - x)];
    }

    // output[:, nx:] = sino[n_a2:, ::-1][:, 2 * d :] = sino[n_a2:, -2*d-1:-n_x-1:-1]
    else { // x in [n_x, n_x2[
        // i in [nx - 2*d - 1, ...,  0] = [nx2 - nx - 1, ..., 0]
        // (n_x2 - 1 - x)  in ]-1, n_x2 - 1 - n_x] = [0, nx - 2d - 1]
        output[y * n_x2 + x] = sinogram[(n_a2 + y)*n_x + (n_x2 - 1 - x)];
    }

}
