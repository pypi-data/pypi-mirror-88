
#include <hip/hip_runtime.h>
/**
    Damping kernel used in the Fourier-Wavelets sinogram destriping method.
*/
__global__ void kern_fourierwavelets(float2* sinoF, int Nx, int Ny, float wsigma) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    int Nfft = Ny/2+1;
    if (gidx >= Nx || gidy >= Nfft) return;

    float m = gidy/wsigma;
    float factor = 1.0f - expf(-(m * m)/2);

    int tid = gidy*Nx + gidx;
    // do not forget the scale factor (here Ny)
    sinoF[tid].x *= factor/Ny;
    sinoF[tid].y *= factor/Ny;
}
