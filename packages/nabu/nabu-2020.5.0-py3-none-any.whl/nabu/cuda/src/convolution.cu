#include "hip/hip_runtime.h"
/*
 * Convolution (without textures)
 * Adapted from OpenCL code of the the silx project
 *
*/

#include "boundary.h"

typedef unsigned int uint;


/******************************************************************************/
/**************************** 1D Convolution **********************************/
/******************************************************************************/


// Convolution with 1D kernel along axis "X" (fast dimension)
// Works for batched 1D on 2D and batched 2D on 3D, along axis "X".
__global__ void convol_1D_X(
    float * input,
    float * output,
    float * filter,
    int L, // filter size
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(L);
    float sum = 0.0f;

    for (int jx = 0; jx <= hR+hL; jx++) {
        CONV_IDX_X; // Get index "x"
        sum += READ_IMAGE_1D_X * filter[L-1 - jx];
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}


// Convolution with 1D kernel along axis "Y"
// Works for batched 1D on 2D and batched 2D on 3D, along axis "Y".
__global__ void convol_1D_Y(
    float * input,
    float * output,
    float * filter,
    int L, // filter size
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(L);
    float sum = 0.0f;

    for (int jy = 0; jy <= hR+hL; jy++) {
        CONV_IDX_Y; // Get index "y"
        sum += READ_IMAGE_1D_Y * filter[L-1 - jy];
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}


// Convolution with 1D kernel along axis "Z"
// Works for batched 1D on 2D and batched 2D on 3D, along axis "Z".
__global__ void convol_1D_Z(
    float * input,
    float * output,
    float * filter,
    int L, // filter size
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(L);
    float sum = 0.0f;

    for (int jz = 0; jz <= hR+hL; jz++) {
        CONV_IDX_Z; // Get index "z"
        sum += READ_IMAGE_1D_Z * filter[L-1 - jz];
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}


/******************************************************************************/
/**************************** 2D Convolution **********************************/
/******************************************************************************/

// Convolution with 2D kernel
// Works for batched 2D on 3D.
__global__ void convol_2D_XY(
    float * input,
    float * output,
    float * filter,
    int Lx, // filter number of columns,
    int Ly, // filter number of rows,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(Lx);
    float sum = 0.0f;

    for (int jy = 0; jy <= hR+hL; jy++) {
        CONV_IDX_Y; // Get index "y"
        for (int jx = 0; jx <= hR+hL; jx++) {
            CONV_IDX_X; // Get index "x"
            sum += READ_IMAGE_2D_XY * filter[(Ly-1-jy)*Lx + (Lx-1 - jx)];
        }
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}


// Convolution with 2D kernel
// Works for batched 2D on 3D.
__global__ void convol_2D_XZ(
    float * input,
    float * output,
    float * filter,
    int Lx, // filter number of columns,
    int Lz, // filter number of rows,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(Lx);
    float sum = 0.0f;

    for (int jz = 0; jz <= hR+hL; jz++) {
        CONV_IDX_Z; // Get index "z"
        for (int jx = 0; jx <= hR+hL; jx++) {
            CONV_IDX_X; // Get index "x"
            sum += READ_IMAGE_2D_XZ * filter[(Lz-1-jz)*Lx + (Lx-1 - jx)];
        }
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}


// Convolution with 2D kernel
// Works for batched 2D on 3D.
__global__ void convol_2D_YZ(
    float * input,
    float * output,
    float * filter,
    int Ly, // filter number of columns,
    int Lz, // filter number of rows,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(Ly);
    float sum = 0.0f;

    for (int jz = 0; jz <= hR+hL; jz++) {
        CONV_IDX_Z; // Get index "z"
        for (int jy = 0; jy <= hR+hL; jy++) {
            CONV_IDX_Y; // Get index "y"
            sum += READ_IMAGE_2D_YZ * filter[(Lz-1-jz)*Ly + (Ly-1 - jy)];
        }
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}



/******************************************************************************/
/**************************** 3D Convolution **********************************/
/******************************************************************************/

// Convolution with 3D kernel
__global__ void convol_3D_XYZ(
    float * input,
    float * output,
    float * filter,
    int Lx, // filter number of columns,
    int Ly, // filter number of rows,
    int Lz, // filter number of rows,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    int c, hL, hR;
    GET_CENTER_HL(Lx);
    float sum = 0.0f;

    for (int jz = 0; jz <= hR+hL; jz++) {
        CONV_IDX_Z; // Get index "z"
        for (int jy = 0; jy <= hR+hL; jy++) {
            CONV_IDX_Y; // Get index "y"
            for (int jx = 0; jx <= hR+hL; jx++) {
                CONV_IDX_X; // Get index "x"
                sum += READ_IMAGE_3D_XYZ * filter[((Lz-1-jz)*Ly + (Ly-1-jy))*Lx + (Lx-1 - jx)];
            }
        }
    }
    output[(gidz*Ny + gidy)*Nx + gidx] = sum;
}

