#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
typedef pycuda::complex<float> complex;



/**
This function padds in-place a 2D array with constant values.
It is designed to leave the data in the "FFT layout", i.e the data is *not*
in the center of the extended/padded data.

In one dimension:

<--------------- N0 ---------------->
|  original data  |  padded values  |
<----- N -------- ><---- Pl+Pr ----->

N0: width of data
Pl, Pr: left/right padding lengths

ASSUMPTIONS:
   - data is already extended before padding (its size is Nx_padded * Ny_padded)
   - the original data lies in the top-left quadrant.


**/
__global__ void padding_constant(
    float* data,
    int Nx,
    int Ny,
    int Nx_padded,
    int Ny_padded,
    int pad_left_len,
    int pad_right_len,
    int pad_top_len,
    int pad_bottom_len,
    float pad_left_val,
    float pad_right_val,
    float pad_top_val,
    float pad_bottom_val
) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((x >= Nx_padded) || (y >= Ny_padded)) return;
    int idx = y*Nx_padded  +  x;

    // data[s0:s0+Pd, :s1] = pad_bottom_val
    if ((Ny <= y) && (y < Ny+pad_bottom_len) && (x < Nx))
        data[idx] = pad_bottom_val;
    // data[s0+Pd:s0+Pd+Pu, :s1] = pad_top_val
    else if ((Ny + pad_bottom_len <= y) && (y < Ny+pad_bottom_len+pad_top_len) && (x < Nx))
        data[idx] = pad_top_val;
    // data[:, s1:s1+Pr] = pad_right_val
    else if ((Nx <= x) && (x < Nx+pad_right_len))
        data[idx] = pad_right_val;
    // data[:, s1+Pr:s1+Pr+Pl] = pad_left_val
    else if ((Nx+pad_right_len <= x) && (x < Nx+pad_right_len+pad_left_len))
        data[idx] = pad_left_val;
    // top-left quadrant
    else
        return;
}



__global__ void padding_edge(
    float* data,
    int Nx,
    int Ny,
    int Nx_padded,
    int Ny_padded,
    int pad_left_len,
    int pad_right_len,
    int pad_top_len,
    int pad_bottom_len
) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((x >= Nx_padded) || (y >= Ny_padded)) return;
    int idx = y*Nx_padded  +  x;

    //
    // This kernel can be optimized:
    //   - Optimize the logic to use less comparisons
    //   - Store the values data[0], data[s0-1, 0], data[0, s1-1], data[s0-1, s1-1]
    //     into shared memory to read only once from global mem.
    //

    // data[s0:s0+Pd, :s1] = data[s0, :s1]
    if ((Ny <= y) && (y < Ny+pad_bottom_len) && (x < Nx))
        data[idx] = data[(Ny-1)*Nx_padded+x];
    // data[s0+Pd:s0+Pd+Pu, :s1] = data[0, :s1]
    else if ((Ny + pad_bottom_len <= y) && (y < Ny+pad_bottom_len+pad_top_len) && (x < Nx))
        data[idx] = data[x];
    // data[:s0, s1:s1+Pr] = data[:s0, s1]
    else if ((y < Ny) && (Nx <= x) && (x < Nx+pad_right_len))
        data[idx] = data[y*Nx_padded + Nx-1];
    // data[:s0, s1+Pr:s1+Pr+Pl] = data[:s0, 0]
    else if ((y < Ny) && (Nx+pad_right_len <= x) && (x < Nx+pad_right_len+pad_left_len))
        data[idx] = data[y*Nx_padded];
    // data[s0:s0+Pb, s1:s1+Pr] = data[s0-1, s1-1]
    else if ((Ny <= y && y < Ny + pad_bottom_len) && (Nx <= x && x < Nx + pad_right_len))
        data[idx] = data[(Ny-1)*Nx_padded + Nx-1];
    // data[s0:s0+Pb, s1+Pr:s1+Pr+Pl] = data[s0-1, 0]
    else if ((Ny <= y && y < Ny + pad_bottom_len) && (Nx+pad_right_len <= x && x < Nx + pad_right_len+pad_left_len))
        data[idx] = data[(Ny-1)*Nx_padded];
    // data[s0+Pb:s0+Pb+Pu, s1:s1+Pr] = data[0, s1-1]
    else if ((Ny+pad_bottom_len <= y && y < Ny + pad_bottom_len+pad_top_len) && (Nx <= x && x < Nx + pad_right_len))
        data[idx] = data[Nx-1];
    // data[s0+Pb:s0+Pb+Pu, s1+Pr:s1+Pr+Pl] = data[0, 0]
    else if ((Ny+pad_bottom_len <= y && y < Ny + pad_bottom_len+pad_top_len) && (Nx+pad_right_len <= x && x < Nx + pad_right_len+pad_left_len))
        data[idx] = data[0];
    // top-left quadrant
    else
        return;
}



/**
 // Old stuff

#define BLOCK_SIZE 16

__global__ void  zeropadding_complex(float* input, int sino_width, complex* output) {

  const int dim_x = gridDim.x * blockDim.x;
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  long out_idx = idy * dim_x + idx;

  int lpart = sino_width/2;
  int rpart = sino_width/2;
  int dim_x2 = dim_x/2;
  int len_to_lpart = dim_x - lpart;

  output[out_idx]._M_re = (idx < rpart) ?
    input[idy * sino_width + (lpart + idx)] :
    (idx < (dim_x2 + (dim_x2 - lpart))) ? 0.0f : input[idy * sino_width + (idx - len_to_lpart)];
  output[out_idx]._M_im = 0.0f;
}

__global__ void  zeropadding_real(float* input, int sino_width, int dim2_fft, float* output) {
  const int dim_x = gridDim.x * blockDim.x;
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  long out_idx = idy * dim2_fft + idx;

  int lpart = sino_width/2;
  int rpart = sino_width/2;
  int dim_x2 = dim_x/2;
  int len_to_lpart = dim_x - lpart;

  output[out_idx] = (idx < rpart) ? input[idy * sino_width + (lpart + idx)] :
    (idx < (dim_x2 + (dim_x2 - lpart))) ? 0.0f : input[idy * sino_width + (idx - len_to_lpart)];
}
**/




