
#include <hip/hip_runtime.h>
typedef unsigned int uint;


__global__ void histogram(
    float * array,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz,  // input/output depth
    float arr_min, // array minimum value
    float arr_max, // array maximum value
    uint* hist,    // histogram
    int nbins      // histogram size (number of bins)
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= Nx) || (gidy >= Ny) || (gidz >= Nz)) return;

    float val = array[(gidz*Ny + gidy)*Nx + gidx];
    float bin_pos = nbins * ((val - arr_min) / (arr_max - arr_min));
    uint bin_left = min((uint) bin_pos, nbins-1);
    atomicAdd(hist + bin_left, 1);
}
