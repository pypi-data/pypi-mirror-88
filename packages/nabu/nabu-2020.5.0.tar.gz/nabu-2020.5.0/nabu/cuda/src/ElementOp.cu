#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>

typedef pycuda::complex<float> complex;

// arr2D *= arr1D (line by line, i.e along fast dim)
__global__ void inplace_complex_mul_2Dby1D(complex* arr2D, complex* arr1D, int width, int height) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((x >= width) || (y >= height)) return;
    // This does not seem to work
    // Use hipCmulf of hip/hip_complex.h ?
    //~ arr2D[y*width + x] *= arr1D[x];
    int i = y*width + x;
    complex a = arr2D[i];
    complex b = arr1D[x];
    arr2D[i]._M_re = a._M_re * b._M_re - a._M_im * b._M_im;
    arr2D[i]._M_im = a._M_im * b._M_re + a._M_re * b._M_im;
}


// arr3D *= arr1D (along fast dim)
__global__ void inplace_complex_mul_3Dby1D(complex* arr3D, complex* arr1D, int width, int height, int depth) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int z = blockDim.z * blockIdx.z + threadIdx.z;
    if ((x >= width) || (y >= height) || (z >= depth)) return;
    // This does not seem to work
    // Use hipCmulf of hip/hip_complex.h ?
    //~ arr3D[(z*height + y)*width + x] *= arr1D[x];
    int i = (z*height + y)*width + x;
    complex a = arr3D[i];
    complex b = arr1D[x];
    arr3D[i]._M_re = a._M_re * b._M_re - a._M_im * b._M_im;
    arr3D[i]._M_im = a._M_im * b._M_re + a._M_re * b._M_im;
}



// arr2D *= arr2D
__global__ void inplace_complex_mul_2Dby2D(complex* arr2D_out, complex* arr2D_other, int width, int height) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((x >= width) || (y >= height)) return;
    int i = y*width + x;
    complex a = arr2D_out[i];
    complex b = arr2D_other[i];
    arr2D_out[i]._M_re = a._M_re * b._M_re - a._M_im * b._M_im;
    arr2D_out[i]._M_im = a._M_im * b._M_re + a._M_re * b._M_im;
}


// arr2D *= arr2D
__global__ void inplace_complexreal_mul_2Dby2D(complex* arr2D_out, float* arr2D_other, int width, int height) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((x >= width) || (y >= height)) return;
    int i = y*width + x;
    complex a = arr2D_out[i];
    float b = arr2D_other[i];
    arr2D_out[i]._M_re *= b;
    arr2D_out[i]._M_im *= b;
}


#ifndef DO_CLIP_MIN
    #define DO_CLIP_MIN 0
#endif

#ifndef DO_CLIP_MAX
    #define DO_CLIP_MAX 0
#endif

// arr = -log(arr)
__global__ void nlog(float* array, int Nx, int Ny, int Nz, float clip_min, float clip_max) {
    uint x = blockDim.x * blockIdx.x + threadIdx.x;
    uint y = blockDim.y * blockIdx.y + threadIdx.y;
    uint z = blockDim.z * blockIdx.z + threadIdx.z;
    if ((x >= Nx) || (y >= Ny) || (z >= Nz)) return;
    uint pos = (z*Ny + y)*Nx + x;
    float val = array[pos];
    #if DO_CLIP_MIN
        val = fmaxf(val, clip_min);
    #endif
    #if DO_CLIP_MAX
        val = fminf(val, clip_max);
    #endif
    array[pos] = -logf(val);
}



// Reverse elements of a 2D array along "x", i.e:
// arr = arr[:, ::-1]
// launched with grid (Nx/2, Ny)
__global__ void reverse2D_x(float* array, int Nx, int Ny) {
    uint x = blockDim.x * blockIdx.x + threadIdx.x;
    uint y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((x >= Nx/2) || (y >= Ny)) return;
    uint pos = y*Nx + x;
    uint pos2 = y*Nx + (Nx - 1 - x);
    float tmp = array[pos];
    array[pos] = array[pos2];
    array[pos2] = tmp;
}

