
#include <hip/hip_runtime.h>
typedef unsigned int uint;


/**
 * Chebyshev background removal.
 * This kernel does a degree 2 polynomial estimation of each line of an array,
 * and then subtracts the estimation from each line.
 * This process is done in-place.
 */
__global__ void normalize_chebyshev(
    float * array,
    int Nx, // input/output number of columns
    int Ny, // input/output number of rows
    int Nz  // input/output depth
)
{
    uint gidx = blockDim.x * blockIdx.x + threadIdx.x;
    uint gidy = blockDim.y * blockIdx.y + threadIdx.y;
    uint gidz = blockDim.z * blockIdx.z + threadIdx.z;
    if ((gidx >= 1) || (gidy >= Ny) || (gidz >= Nz)) return;

    float ff0=0.0f, ff1=0.0f, ff2=0.0f;
    float sum0=0.0f, sum1=0.0f, sum2=0.0f;
    float f0, f1, f2, x;
    for (int j=0; j < Nx; j++) {
        uint pos = (gidz*Ny + gidy)*Nx + j;
        float arr_val = array[pos];
        x = 2.0f*(j + 0.5f - Nx/2.0f)/Nx;
        f0 = 1.0f;
        f1 = x;
        f2 = (3.0f*x*x-1.0f);
        ff0 = ff0 + f0 * arr_val;
        ff1 = ff1 + f1 * arr_val;
        ff2 = ff2 + f2 * arr_val;
        sum0 += f0 * f0;
        sum1 += f1 * f1;
        sum2 += f2 * f2;
    }
    for (int j=0; j< Nx; j++) {
        uint pos = (gidz*Ny + gidy)*Nx + j;
        x = 2.0f*(j+0.5f-Nx/2.0f)/Nx;
        f0 = 1.0f;
        f1 = x;
        f2 = (3.0f*x*x-1.0f);
        array[pos] -= ff0*f0/sum0 + ff1*f1/sum1 + ff2*f2/sum2;
    }
}
