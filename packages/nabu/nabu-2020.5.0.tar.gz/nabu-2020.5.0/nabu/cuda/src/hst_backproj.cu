#include "hip/hip_runtime.h"
texture<float, 2, hipReadModeElementType> texProj;
//~ hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();

__global__ void backproj(
    int num_proj,
    int num_bins,
    float axis_position,
    float *d_SLICE,
    float gpu_offset_x,
    float gpu_offset_y,
    float * d_cos_s,
    float * d_sin_s,
    float *  d_axis_s)
{
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int tidy = threadIdx.y;
    const int bidy = blockIdx.y;

    __shared__  float   shared[768];
    float  * sh_sin  = shared;
    float  * sh_cos  = shared+256;
    float  * sh_axis = sh_cos+256;

    float pcos, psin;
    float h0, h1, h2, h3;

    const float apos_off_x = gpu_offset_x - axis_position ;
    const float apos_off_y = gpu_offset_y - axis_position ;
    float acorr05;
    float res0 = 0, res1 = 0, res2 = 0, res3 = 0;
    const float bx00 = (32 * bidx + 2 * tidx + apos_off_x);
    const float by00 = (32 * bidy + 2 * tidy + apos_off_y);

    int read=0;
    for (int proj=0; proj<num_proj; proj++) {
        if(proj>=read) {
            __syncthreads();
            int ip = tidy*16+tidx;
            if( read+ip < num_proj) {
                sh_cos [ip] = d_cos_s[read+ip];
                sh_sin [ip] = d_sin_s[read+ip];
                sh_axis[ip] = d_axis_s[read+ip];
            }
            read = read + 256; // 256=16*16 block size
            __syncthreads();
        }
        pcos = sh_cos[256 - read + proj] ;
        psin = sh_sin[256 - read + proj] ;
        acorr05 = sh_axis[256 - read + proj];

        h0 =  acorr05 + bx00*pcos - by00*psin;
        h1 =  acorr05 + bx00*pcos - (by00+1)*psin;
        h2 =  acorr05 + (bx00+1)*pcos - by00*psin;
        h3 =  acorr05 + (bx00+1)*pcos - (by00+1)*psin;

        if(h0 >= 0 && h0 < num_bins) res0 += tex2D(texProj, h0 + 0.5f, proj + 0.5f);
        if(h1>=0 && h1<num_bins) res1 += tex2D(texProj, h1 +0.5f, proj + 0.5f);
        if(h2>=0 && h2<num_bins) res2 += tex2D(texProj, h2 +0.5f, proj + 0.5f);
        if(h3>=0 && h3<num_bins) res3 += tex2D(texProj, h3 +0.5f, proj + 0.5f);
    }
    d_SLICE[32*gridDim.x * (bidy*32+tidy*2+0) + bidx*32 + tidx*2 + 0] = res0;
    d_SLICE[32*gridDim.x * (bidy*32+tidy*2+1) + bidx*32 + tidx*2 + 0] = res1;
    d_SLICE[32*gridDim.x * (bidy*32+tidy*2+0) + bidx*32 + tidx*2 + 1] = res2;
    d_SLICE[32*gridDim.x * (bidy*32+tidy*2+1) + bidx*32 + tidx*2 + 1] = res3;
}



