
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#define BLOCK_SIZE 16

__global__ void dfi_cuda_swap_quadrants_complex(hipfftComplex *input, hipfftComplex *output, int dim_x) {

    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    const int dim_y = gridDim.y * blockDim.y; //a half of real length

    output[idy * dim_x + idx] = input[(dim_y + idy) * dim_x + idx + 1];
    output[(dim_y + idy) * dim_x + idx] = input[idy * dim_x + idx + 1];
}

__global__ void dfi_cuda_swap_quadrants_real(hipfftReal *output) {

    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    const int dim_x = gridDim.x * blockDim.x;
    int dim_x2 = dim_x/2, dim_y2 = dim_x2;
    long sw_idx1, sw_idx2;

    sw_idx1 = idy * dim_x + idx;

    hipfftReal temp = output[sw_idx1];
    if (idx < dim_x2) {
        sw_idx2 = (dim_y2 + idy) * dim_x + (dim_x2 + idx);
        output[sw_idx1] = output[sw_idx2];
        output[sw_idx2] = temp;
    }
    else {
        sw_idx2 = (dim_y2 + idy) * dim_x + (idx - dim_x2);
        output[sw_idx1] = output[sw_idx2];
        output[sw_idx2] = temp;
    }
}

__global__ void swap_full_quadrants_complex(hipfftComplex *output) {

    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    const int dim_x = gridDim.x * blockDim.x;
    int dim_x2 = dim_x/2, dim_y2 = dim_x2;
    long sw_idx1, sw_idx2;

    sw_idx1 = idy * dim_x + idx;

    hipfftComplex temp = output[sw_idx1];
    if (idx < dim_x2) {
        sw_idx2 = (dim_y2 + idy) * dim_x + (dim_x2 + idx);
        output[sw_idx1] = output[sw_idx2];
        output[sw_idx2] = temp;
    }
    else {
        sw_idx2 = (dim_y2 + idy) * dim_x + (idx - dim_x2);
        output[sw_idx1] = output[sw_idx2];
        output[sw_idx2] = temp;
    }
}

__global__ void dfi_cuda_crop_roi(hipfftReal *input, int x, int y, int roi_x, int roi_y, int raster_size, float scale, hipfftReal *output) {

    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if (idx < roi_x && idy < roi_y) {
        output[idy * roi_x + idx] = input[(idy + y) * raster_size + (idx + x)] * scale;
    }
}
