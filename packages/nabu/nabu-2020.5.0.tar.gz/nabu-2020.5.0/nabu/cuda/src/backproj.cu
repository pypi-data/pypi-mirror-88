#include "hip/hip_runtime.h"
#ifndef SHARED_SIZE
    #define SHARED_SIZE 256
#endif

texture<float, 2, hipReadModeElementType> tex_projections;

#ifdef BACKPROJ3D
texture<float, 3, hipReadModeElementType> tex_projections3D;
#endif

/**

Implementation details
-----------------------
This implementation uses two pre-computed arrays in global memory:
  cos(theta)  -> d_cos
  -sin(theta) -> d_msin
As the backprojection is voxel-driven, each thread will, at some point,
need cos(theta) and -sin(theta) for *all* theta.
Thus, we need to pre-fetch d_cos and d_msin in the fastest cached memory.
Here we use the shared memory (faster than constant memory and texture).
Each thread group will pre-fetch values from d_cos and d_msin to shared memory
Initially, we fetched as much values as possible, ending up in a block of 1024
threads (32, 32). However, it turns out that performances are best with (16, 16)
blocks.
**/

// Backproject one sinogram
// One thread handles up to 4 pixels in the output slice
// the case num_projs > 1024 has to be included.
__global__ void backproj(
    float* d_slice,
    int num_projs,
    int num_bins,
    #ifdef BACKPROJ3D
    int num_slices,
    #endif
    float axis_position,
    int n_x,
    int n_y,
    int offset_x,
    int offset_y,
    float* d_cos,
    float* d_msin,
    #ifdef DO_AXIS_CORRECTION
    float* d_axis_corr,
    #endif
    float scale_factor
)
{
    int x = offset_x + blockDim.x * blockIdx.x + threadIdx.x;
    int y = offset_y + blockDim.y * blockIdx.y + threadIdx.y;
    #ifdef BACKPROJ3D
    int z = blockDim.z * blockIdx.z + threadIdx.z;
    #endif

    int Gx = blockDim.x * gridDim.x;
    int Gy = blockDim.y * gridDim.y;

    // (xr, yr)    (xrp, yr)
    // (xr, yrp)   (xrp, yrp)
    float xr = x - axis_position, yr = y - axis_position;
    float xrp = xr + Gx, yrp = yr + Gy;

    /*volatile*/ __shared__ float s_cos[SHARED_SIZE];
    /*volatile*/ __shared__ float s_msin[SHARED_SIZE];
    #ifdef DO_AXIS_CORRECTION
    /*volatile*/ __shared__ float s_axis[SHARED_SIZE];
    float axcorr;
    #endif

    int next_fetch = 0;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    float costheta, msintheta;
    float h1, h2, h3, h4;
    float sum1 = 0.0f, sum2 = 0.0f, sum3 = 0.0f, sum4 = 0.0f;

    for (int proj = 0; proj < num_projs; proj++) {
        if (proj == next_fetch) {
            // Fetch SHARED_SIZE values to shared memory
            __syncthreads();
            if (next_fetch + tid < num_projs) {
                s_cos[tid] = d_cos[next_fetch + tid];
                s_msin[tid] = d_msin[next_fetch + tid];
                #ifdef DO_AXIS_CORRECTION
                s_axis[tid] = d_axis_corr[next_fetch + tid];
                #endif
            }
            next_fetch += SHARED_SIZE;
            __syncthreads();
        }

        costheta = s_cos[proj - (next_fetch - SHARED_SIZE)];
        msintheta = s_msin[proj - (next_fetch - SHARED_SIZE)];
        #ifdef DO_AXIS_CORRECTION
        axcorr = s_axis[proj - (next_fetch - SHARED_SIZE)];
        #endif
        float c1 = fmaf(costheta, xr, axis_position); // cos(theta)*xr + axis_pos
        float c2 = fmaf(costheta, xrp, axis_position); // cos(theta)*(xr + Gx) + axis_pos
        float s1 = fmaf(msintheta, yr, 0.0f); // -sin(theta)*yr
        float s2 = fmaf(msintheta, yrp, 0.0f); // -sin(theta)*(yr + Gy)
        h1 = c1 + s1;
        h2 = c2 + s1;
        h3 = c1 + s2;
        h4 = c2 + s2;
        #ifdef DO_AXIS_CORRECTION
        h1 += axcorr;
        h2 += axcorr;
        h3 += axcorr;
        h4 += axcorr;
        #endif

        #ifdef BACKPROJ3D
        if (h1 >= 0 && h1 < num_bins) sum1 += tex3D(tex_projections3D, h1 + 0.5f, proj + 0.5f, z + 0.5f);
        if (h2 >= 0 && h2 < num_bins) sum2 += tex3D(tex_projections3D, h2 + 0.5f, proj + 0.5f, z + 0.5f);
        if (h3 >= 0 && h3 < num_bins) sum3 += tex3D(tex_projections3D, h3 + 0.5f, proj + 0.5f, z + 0.5f);
        if (h4 >= 0 && h4 < num_bins) sum4 += tex3D(tex_projections3D, h4 + 0.5f, proj + 0.5f, z + 0.5f);
        #else
        if (h1 >= 0 && h1 < num_bins) sum1 += tex2D(tex_projections, h1 + 0.5f, proj + 0.5f);
        if (h2 >= 0 && h2 < num_bins) sum2 += tex2D(tex_projections, h2 + 0.5f, proj + 0.5f);
        if (h3 >= 0 && h3 < num_bins) sum3 += tex2D(tex_projections, h3 + 0.5f, proj + 0.5f);
        if (h4 >= 0 && h4 < num_bins) sum4 += tex2D(tex_projections, h4 + 0.5f, proj + 0.5f);
        #endif
    }

    x -= offset_x;
    y -= offset_y;

    // useful only if n_x < blocksize_x or n_y < blocksize_y
    if (x >= n_x) return;
    if (y >= n_y) return;

    #ifdef BACKPROJ3D
    d_slice[(z*n_y + y)*(n_x) + x] = sum1 * scale_factor;
    if (Gx + x < n_x)
        d_slice[(z*n_y + y)*(n_x) + Gx + x] = sum2 * scale_factor;
    if (Gy + y < n_y) {
        d_slice[(z*n_y + y + Gy)*(n_x) + x] = sum3 * scale_factor;
    if (Gx + x < n_x)
        d_slice[(z*n_y + y + Gy)*(n_x) + Gx + x] = sum4 * scale_factor;
    }
    #else
    d_slice[y*(n_x) + x] = sum1 * scale_factor;
    if (Gx + x < n_x)
        d_slice[y*(n_x) + Gx + x] = sum2 * scale_factor;
    if (Gy + y < n_y) {
        d_slice[(y+Gy)*(n_x) + x] = sum3 * scale_factor;
    if (Gx + x < n_x)
        d_slice[(y+Gy)*(n_x) + Gx + x] = sum4 * scale_factor;
    }
    #endif
}
